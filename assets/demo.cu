#include "hip/hip_runtime.h"
#include "x.hpp"
#include "x-mat.hpp"

namespace liong {

extern "C" __constant__ LaunchConfig cfg;

extern "C" __global__ void __closesthit__() {
  auto mat = GET_MATERIAL_PTR(HitMaterial);
  auto pColor = (uint32_t*)WORDS2PTR(optixGetPayload_0(), optixGetPayload_1());
  *pColor = 0xFFFF00FF;
}

extern "C" __global__ void __anyhit__() {
}

extern "C" __global__ void __miss__() {
  auto mat = GET_MATERIAL_PTR(MissMaterial);
  auto pColor = (uint32_t*)WORDS2PTR(optixGetPayload_0(), optixGetPayload_1());
  *pColor = 0xFF000000;
}

extern "C" __global__ void __raygen__() {
  auto x = optixGetLaunchIndex().x;
  auto y = optixGetLaunchIndex().y;
  auto u = ((float)(x * 2 + 1) / cfg.width - 1);
  auto v = ((float)(y * 2 + 1) / cfg.height - 1);
  auto i = x + y * cfg.width;
  uint32_t color = 0;
  uint32_t wColor[] = PTR2WORDS(&color);

  optixTrace(cfg.trav, { u, v, 1.0 }, { 0.0, 0.0, -1.0 }, 0.f, 1e20f, 0.0f,
    OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 1, 0,
    wColor[0], wColor[1]);

  cfg.framebuf[i] = color;
}

}
